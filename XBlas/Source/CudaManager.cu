#include "../Include/CudaManager.cuh"
#include<system_error>
#include<string>

namespace XBlas
{
	CudaManager::CudaManager()
	{
		hipblasStatus_t status = hipblasCreate(&handle);
		if (status != HIPBLAS_STATUS_SUCCESS)
		{
			std::string message = "CUBLAS initialization failed with error code " + status;
			throw std::runtime_error(message);
		}
	}

	CudaManager::~CudaManager()
	{
		hipblasDestroy(handle);
	}
}