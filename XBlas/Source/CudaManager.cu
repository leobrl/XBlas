#include "../Include/CudaManager.cuh"
#include<system_error>
#include<string>

namespace XBlas
{
	CudaManager::CudaManager()
	{
		hipblasStatus_t hipblasStatus_t = hipblasCreate(&handle);
		if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
		{
			std::string message = "CUBLAS initialization failed with error code " + hipblasStatus_t;
			throw std::runtime_error(message);
		}

		hipsolverStatus_t cusolverStatus = hipsolverDnCreate(&solver_handle);
		if (cusolverStatus != HIPSOLVER_STATUS_SUCCESS)
		{
			std::string message = "CUBLAS initialization failed with error code " + cusolverStatus;
			throw std::runtime_error(message);
		}
	}

	CudaManager::~CudaManager()
	{
		hipblasDestroy(handle);
	}
}